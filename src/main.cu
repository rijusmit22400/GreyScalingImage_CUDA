#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <string>

#include "stb_image.h"
#include "stb_image_write.h"

using namespace std;

typedef struct Pixel{
    unsigned char r, g, b, a;
} Pixel;

void ConvertImageToGrayCPU(char* imageRGBA, int width, int height){
    for(int i=0;i<height;i++){
        for(int j=0;j<width;j++){
            Pixel* pixel = (Pixel*)&imageRGBA[(i*width)*4+4*j];
            float pixelValue = pixel->r*0.299f + pixel->g*0.587f + pixel->b*0.114f;
            unsigned char gray = (unsigned char)pixelValue;
            pixel->r = gray;
            pixel->g = gray;
            pixel->b = gray;
            pixel->a = 255;
        }
    }
} 

int main(){
    // Hardcoded image filename
    const char* filename = "sample_data_color.jpg";

    // Open image
    int width, height, componentCount;
    unsigned char* imageData = stbi_load(filename, &width, &height, &componentCount, 0);
    if(!imageData){
        cout << "Failed to open \"" << filename << "\"\n";
        return -1;
    }

    // Validate image size
    if(width % 32 != 0 || height % 32 != 0){
        // Image size must be a multiple of 32
        cout << "Image size must be a multiple of 32\n";
        stbi_image_free(imageData);
        return -1;
    }

    cout << "Image size: " << width << "x" << height << "\n";
    cout << "Loading....." << "\n";
    cout << "Processing the image....." << "\n";
    ConvertImageToGrayCPU((char*)imageData, width, height);
    cout << "DONE" << "\n"; 

    // Building output filename
    string outputFilename = filename;
    size_t dotPos = outputFilename.find_last_of('.');
    if (dotPos != string::npos) {
        outputFilename = outputFilename.substr(0, dotPos) + "_gray.jpg";
    } else {
        outputFilename += "_gray.jpg";
    }

    // Write image back
    stbi_write_jpg(outputFilename.c_str(), width, height, 4, imageData, 100);

    // Close 
    stbi_image_free(imageData);

    cout << "Converted image saved as \"" << outputFilename << "\"\n";

    // Random cout statements for debugging
    cout << "Debug statement 1\n";
    cout << "Debug statement 2\n";
    cout << "Debug statement 3\n";

    return 0;
}
