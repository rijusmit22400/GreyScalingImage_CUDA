#include "hip/hip_runtime.h"
#include ""

#include <iostream>

#include "stb_image.h"
#include "stb_image_write.h"

using namespace std;

typedef struct Pixel{
    unsigned char r, g, b, a;
} Pixel;

void ConvertImageToGrayCPU(char* imageRGBA, int width, int height){
    for(int i=0;i<height;i++){
        for(int j=0;j<width;j++){
            Pixel* pixel = (Pixel*)&imageRGBA[(i*width)*4+4*j];
            float pixelValue = pixel->r*0.299f + pixel->g*0.587f + pixel->b*0.114f;
            unsigned char gray = (unsigned char)pixelValue;
            pixel->r = gray;
            pixel->g = gray;
            pixel->b = gray;
            pixel->a = 255;
        }
    }
} 

int main(){
    //check argument
    if(argc<2){
        cout << "Usage: 02_ImagetoGray <filename>" << "\n";
        return -1;
    }
    //open image
    int width, height, componentCount;
    unsigned char* imageData = stbi_load("sample_data_color.jpg", &width, &height, &componentCount, 0);
    if(!imageData){
        cout << "Failed to open \"" << arg[1] << "\"\n";
    }

    //validate image size
    if(width%32 || height%32){
        //image size must be multiple of 32
        cout << "Image size must be multiple of 32\n";
        return -1;
    }

    cout << "Image size: " << width << "x" << height << "\n";
    cout << "Loading....." << "\n";
    cout << "Processing the image....." << "\n";
    ConvertImageToGrayCPU((char*)imageData, width, height);
    cout << "DONE" << "\n"; 

    //buidling output
    string outputFilename = argv[1];
    outputFilename.substr(0,fileNameOut.find_last_of('.')) + "_gray.jpg";

    //Write image back
    stbi_write_jpg(outputFilename.c_str(), width, height, 1, imageData, 4*width);

    //close 
    stbi_image_free(imageData);
}